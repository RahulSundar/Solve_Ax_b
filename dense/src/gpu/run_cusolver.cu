//Ref: https://developer.nvidia.com/sites/default/files/akamai/cuda/files/Misc/mygpu.pdf
//REF: https://github.com/NVIDIA/cuda-samples/blob/master/Samples/cuSolverDn_LinearSolver/cuSolverDn_LinearSolver.cpp
#include <ctime>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <iostream>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>
#define BILLION 1000000000L ;
#include "run_cusolver.h"
#include "hip/hip_runtime_api.h"


void GPU_solver::run_cuda_gensolver(const int &nn, const std::vector<double> &Aex,
		const std::vector<double> &bex) {
    struct timespec start, stop;
    int n=nn, lda;
    lda = n;
    double accum ; // elapsed time variable
    hipblasStatus_t stat ;
    hipError_t cudaStatus ;
    hipsolverStatus_t cusolverStatus ;
    hipsolverHandle_t handle ;
    double *d_A, *d_b, *d_Work; // Device memory, coeff .matrix, rhs, workspace
    int *d_pivot, *d_info, Lwork; // pivots, info, worksp. size
    int info_gpu = 0;
    float free_m,total_m,used_m;
    size_t free_t,total_t;

    // prepare memory on the host
    h_A_.resize(n*n);
    h_b_.resize(n);
    h_A_ = Aex;
    h_b_ = bex;
    cudaStatus = hipGetDevice(0);
    //checkCudaErrors(cudaStatus); // this yields an error ?
    cusolverStatus = hipsolverDnCreate (&handle );
    // hipsolverDnCreate seems to conflict with thrust::device_memory
    // 0118-2020 
    // prepare memory on the device
    checkCudaErrors(hipMalloc(( void **)&d_A,     n*n* sizeof (double)));
    checkCudaErrors(hipMalloc(( void **)&d_b,     n*   sizeof (double)));
    checkCudaErrors(hipMalloc(( void **)&d_pivot, n*   sizeof (int)));
    checkCudaErrors(hipMalloc(( void **)&d_info,       sizeof (int )));
    checkCudaErrors(hipMemcpy(d_A,h_A_.data(),n*n*sizeof(double),
                    hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_b,h_b_.data(),n*  sizeof(double),
                    hipMemcpyHostToDevice));
    //vvvvvvvvvvvvvvvvvvvvvvvv
    checkCudaErrors(hipsolverDnDgetrf_bufferSize(handle,n,n,d_A,lda,&Lwork)); 
    checkCudaErrors(hipMalloc (( void **)&d_Work , Lwork * sizeof (double)));
    clock_gettime ( CLOCK_REALTIME ,&start ); // timer start
    checkCudaErrors(hipsolverDnDgetrf(handle,n,n,d_A,lda,d_Work,d_pivot,d_info));
    checkCudaErrors(hipsolverDnDgetrs(handle, HIPBLAS_OP_N, n, 1, d_A, lda,
                    d_pivot, d_b, n, d_info));
    checkCudaErrors(hipDeviceSynchronize());
    //^^^^^^^^^^^^^^^^^^
    clock_gettime ( CLOCK_REALTIME ,&stop ); // timer stop
    accum =( stop.tv_sec - start.tv_sec )+ // elapsed time
	( stop.tv_nsec - start.tv_nsec )/( double ) BILLION ;
    printf (" getrf + getrs time : %lf sec .\n",accum ); // print el. time
    checkCudaErrors(hipMemcpy (&info_gpu, d_info, sizeof (int),
                    hipMemcpyDeviceToHost )); // d_info -> info_gpu
    //printf (" after getrf + getrs : info_gpu = %d\n", info_gpu );
    checkCudaErrors(hipMemcpy (h_b_.data(), d_b , n* sizeof (double) ,
                    hipMemcpyDeviceToHost)); //
    hipMemGetInfo(&free_t,&total_t);
    free_m =(uint)free_t/1024./1024. ;
    total_m=(uint)total_t/1024./1024.;
    used_m=total_m-free_m;
    std::cout << "Used GPU mem=" << used_m << " MB. Free GPU mem=" << free_m << " MB\n";
    // free memory
    checkCudaErrors(hipFree (d_A));
    checkCudaErrors(hipFree (d_b));
    checkCudaErrors(hipFree (d_pivot));
    checkCudaErrors(hipFree (d_info));
    checkCudaErrors(hipFree (d_Work));
    cusolverStatus = hipsolverDnDestroy (handle);
    cudaStatus = hipDeviceReset ();
    checkCudaErrors(cudaStatus);
}

void GPU_solver::run_cuda_symsolver(const int &nn, const std::vector<double> &Aex,
        const std::vector<double> &bex) {
    struct timespec start, stop;
    int n=nn, lda;
    lda = n;
    double accum ; // elapsed time variable
    hipblasStatus_t stat ;
    hipError_t cudaStatus ;
    hipsolverStatus_t cusolverStatus ;
    hipsolverHandle_t handle ;
    double *d_A, *d_b, *d_Work; // Device memory, coeff .matrix, rhs, workspace
    int *d_pivot, *d_info, Lwork; // pivots, info, worksp. size
    int info_gpu = 0;
    float free_m,total_m,used_m;
    size_t free_t,total_t;
    // prepare memory on the host
    h_A_.resize(n*n);
    h_b_.resize(n);
    h_A_ = Aex;
    h_b_ = bex;
    cudaStatus = hipGetDevice(0);
    //checkCudaErrors(cudaStatus); // this yields an error ?
    cusolverStatus = hipsolverDnCreate (&handle );
    // hipsolverDnCreate seems to conflict with thrust::device_memory
    // 0118-2020
    // prepare memory on the device
    checkCudaErrors(hipMalloc(( void **)&d_A,     n*n* sizeof (double)));
    checkCudaErrors(hipMalloc(( void **)&d_b,     n*   sizeof (double)));
    checkCudaErrors(hipMalloc(( void **)&d_pivot, n*   sizeof (int)));
    checkCudaErrors(hipMalloc(( void **)&d_info,       sizeof (int )));
    checkCudaErrors(hipMemcpy(d_A,h_A_.data(),n*n*sizeof(double),
                    hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_b,h_b_.data(),n*  sizeof(double),
                    hipMemcpyHostToDevice));
    //vvvvvvvvvvvvvvvvvvvvvvvv
    checkCudaErrors(hipsolverDnDpotrf_bufferSize(handle, HIPBLAS_FILL_MODE_LOWER,
            n, d_A, lda, &Lwork));
    checkCudaErrors(hipMalloc (( void **)&d_Work , Lwork * sizeof (double)));
    clock_gettime ( CLOCK_REALTIME ,&start ); // timer start
    checkCudaErrors(hipsolverDnDpotrf(handle, HIPBLAS_FILL_MODE_LOWER, n, d_A,
            lda, d_Work, Lwork, d_info));
    checkCudaErrors(hipsolverDnDpotrs(handle, HIPBLAS_FILL_MODE_LOWER, n, 1,
            d_A, lda, d_b, lda, d_info));
    checkCudaErrors(hipDeviceSynchronize());
    //^^^^^^^^^^^^^^^^^^
    clock_gettime ( CLOCK_REALTIME ,&stop ); // timer stop
    accum =( stop.tv_sec - start.tv_sec )+ // elapsed time
    ( stop.tv_nsec - start.tv_nsec )/( double ) BILLION ;
    printf (" getrf + getrs time : %lf sec .\n",accum ); // print el. time
    checkCudaErrors(hipMemcpy (&info_gpu, d_info, sizeof (int),
                    hipMemcpyDeviceToHost )); // d_info -> info_gpu
    //printf (" after getrf + getrs : info_gpu = %d\n", info_gpu );
    checkCudaErrors(hipMemcpy (h_b_.data(), d_b , n* sizeof (double) ,
                    hipMemcpyDeviceToHost)); //
    hipMemGetInfo(&free_t,&total_t);
    free_m =(uint)free_t/1024./1024. ;
    total_m=(uint)total_t/1024./1024.;
    used_m=total_m-free_m;
    std::cout << "Used GPU mem=" << used_m << " MB. Free GPU mem=" << free_m << " MB\n";
    // free memory
    checkCudaErrors(hipFree (d_A));
    checkCudaErrors(hipFree (d_b));
    checkCudaErrors(hipFree (d_pivot));
    checkCudaErrors(hipFree (d_info));
    checkCudaErrors(hipFree (d_Work));
    cusolverStatus = hipsolverDnDestroy (handle);
    cudaStatus = hipDeviceReset ();
    checkCudaErrors(cudaStatus);
}

void GPU_solver::deliver_result(std::vector<double> &x){
    x = h_b_;
}
